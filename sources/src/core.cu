#include "hip/hip_runtime.h"
#include "core.h"

static __constant__ float const_mem[(64 << 10) / sizeof(float)];

struct WuKTimer
{
	hipEvent_t beg, end;
	WuKTimer()
	{
		hipEventCreate(&beg);
		hipEventCreate(&end);
		hipEventRecord(beg);
	}
	~WuKTimer()
	{
		hipEventRecord(end);
		hipEventSynchronize(beg);
		hipEventSynchronize(end);
		float elapsed_time;
		hipEventElapsedTime(
			&elapsed_time,
			beg,
			end);
		printf("%f\n", elapsed_time);
	}
};
namespace v0
{
	static void cudaCallback(
		int k,
		int m,
		int n,
		float *searchPoints,
		float *referencePoints,
		int **results)
	{

		int *tmp = (int *)malloc(sizeof(int) * m);
		int minIndex;
		float minSquareSum, diff, squareSum;

		// Iterate over all search points
		for (int mInd = 0; mInd < m; mInd++)
		{
			minSquareSum = -1;
			// Iterate over all reference points
			for (int nInd = 0; nInd < n; nInd++)
			{
				squareSum = 0;
				for (int kInd = 0; kInd < k; kInd++)
				{
					diff = searchPoints[k * mInd + kInd] - referencePoints[k * nInd + kInd];
					squareSum += (diff * diff);
				}
				if (minSquareSum < 0 || squareSum < minSquareSum)
				{
					minSquareSum = squareSum;
					minIndex = nInd;
				}
			}
			tmp[mInd] = minIndex;
		}

		*results = tmp;
		// Note that you don't have to free searchPoints, referencePoints, and
		// *results by yourself
	}
} // namespace v0
namespace v1
{
	static __global__ void
	get_dis_kernel(
		const int k,
		const int m,
		const int n,
		const float *__restrict__ searchPoints,
		const float *__restrict__ referencePoints,
		float *__restrict__ dis)
	{
		const int
			nInd = threadIdx.x + blockIdx.x * blockDim.x,
			mInd = threadIdx.y + blockIdx.y * blockDim.y;
		if (nInd < n && mInd < m)
		{
			float ans = 0;
			for (int kInd = 0; kInd < k; ++kInd)
			{
				const float d = searchPoints[kInd + mInd * k] - referencePoints[kInd + nInd * k];
				ans += d * d;
			}
			dis[nInd + mInd * n] = ans;
		}
	}
	static void cudaCallback(
		int k,
		int m,
		int n,
		float *searchPoints,
		float *referencePoints,
		int **results)
	{
		thrust::device_vector<float> dis_d(m * n);
		{
			thrust::device_vector<float>
				s_d(searchPoints, searchPoints + k * m),
				r_d(referencePoints, referencePoints + k * n);
			const int BLOCK_DIM_X = 32, BLOCK_DIM_Y = 32;
			//WuKTimer t1;
			get_dis_kernel<<<
				dim3(divup(n, BLOCK_DIM_X), divup(m, BLOCK_DIM_Y)),
				dim3(BLOCK_DIM_X, BLOCK_DIM_Y)>>>(
				k,
				m,
				n,
				thrust::raw_pointer_cast(s_d.data()),
				thrust::raw_pointer_cast(r_d.data()),
				thrust::raw_pointer_cast(dis_d.data()));
		}
		*results = (int *)malloc(sizeof(int) * m);
		{
			//WuKTimer t2;
			for (int i = 0; i < m; ++i)
				(*results)[i] = thrust::min_element(dis_d.begin() + n * i, dis_d.begin() + n * i + n) - dis_d.begin() - n * i;
		}
	}
}; // namespace v1
namespace v2
{
	__global__ void static get_dis_kernel(
		const int k,
		const int m,
		const int n,
		const float *__restrict__ searchPoints,
		const float *__restrict__ referencePoints,
		float *__restrict__ dis)
	{
		const int
			nInd = threadIdx.x + blockIdx.x * blockDim.x,
			mInd = threadIdx.y + blockIdx.y * blockDim.y;
		if (nInd < n && mInd < m)
		{
			float ans = 0;
			for (int kInd = 0; kInd < k; ++kInd)
			{
				const float d = searchPoints[kInd + mInd * k] - referencePoints[kInd + nInd * k];
				ans += d * d;
			}
			dis[nInd + mInd * n] = ans;
		}
	}
	template <int BLOCK_DIM_X>
	static __global__ void
	get_min_kernel(
		const int result_size,
		const int m,
		const int n,
		const float *__restrict__ dis,
		int *__restrict__ result)
	{
		const int ans_id = blockIdx.x + blockIdx.y * gridDim.x;
		if (ans_id >= result_size)
			return;
		__shared__ float dis_s[BLOCK_DIM_X];
		__shared__ int ind_s[BLOCK_DIM_X];
		dis_s[threadIdx.x] = INFINITY;
		for (int nInd = threadIdx.x + blockIdx.x * BLOCK_DIM_X;
			 nInd < n;
			 nInd += gridDim.x * BLOCK_DIM_X)
		{
			const float d = dis[nInd + blockIdx.y * n];
			if (dis_s[threadIdx.x] > d)
			{
				dis_s[threadIdx.x] = d;
				ind_s[threadIdx.x] = nInd;
			}
		}
		__syncthreads();
		for (int offset = BLOCK_DIM_X >> 1; offset > 0; offset >>= 1)
		{
			if (threadIdx.x < offset)
				if (dis_s[threadIdx.x] > dis_s[threadIdx.x ^ offset])
				{
					dis_s[threadIdx.x] = dis_s[threadIdx.x ^ offset];
					ind_s[threadIdx.x] = ind_s[threadIdx.x ^ offset];
				}
			__syncthreads();
		}
		if (threadIdx.x == 0)
			result[ans_id] = ind_s[0];
	}
	static void cudaCallback(
		int k,
		int m,
		int n,
		float *searchPoints,
		float *referencePoints,
		int **results)
	{
		thrust::device_vector<float> dis_d(m * n);
		{
			thrust::device_vector<float>
				s_d(searchPoints, searchPoints + k * m),
				r_d(referencePoints, referencePoints + k * n);
			const int BLOCK_DIM_X = 32, BLOCK_DIM_Y = 32;
			//WuKTimer t1;
			get_dis_kernel<<<
				dim3(divup(n, BLOCK_DIM_X), divup(m, BLOCK_DIM_Y)),
				dim3(BLOCK_DIM_X, BLOCK_DIM_Y)>>>(
				k,
				m,
				n,
				thrust::raw_pointer_cast(s_d.data()),
				thrust::raw_pointer_cast(r_d.data()),
				thrust::raw_pointer_cast(dis_d.data()));
		}
		thrust::device_vector<int> results_d(m);
		{
			const int BLOCK_DIM_X = 1024;
			//WuKTimer t2;
			get_min_kernel<
				BLOCK_DIM_X><<<
				dim3(results_d.size() / m, m),
				BLOCK_DIM_X>>>(
				results_d.size(),
				m,
				n,
				thrust::raw_pointer_cast(dis_d.data()),
				thrust::raw_pointer_cast(results_d.data()));
		}
		thrust::copy(
			results_d.begin(),
			results_d.begin() + m,
			*results = (int *)malloc(sizeof(int) * m));
	}
}; // namespace v2
namespace v3
{
	template <int BLOCK_DIM_X>
	static __global__ void
	cudaCallbackKernel(
		const int k,
		const int m,
		const int n,
		const int result_size,
		const float *__restrict__ searchPoints,
		const float *__restrict__ referencePoints,
		int *__restrict__ result)
	{
		const int ans_id = blockIdx.x + blockIdx.y * gridDim.x;
		if (ans_id >= result_size)
			return;
		__shared__ float dis_s[BLOCK_DIM_X];
		__shared__ int ind_s[BLOCK_DIM_X];
		dis_s[threadIdx.x] = INFINITY;
		for (int mInd = blockIdx.y, nInd = threadIdx.x + blockIdx.x * BLOCK_DIM_X;
			 nInd < n;
			 nInd += gridDim.x * BLOCK_DIM_X)
		{
			float dis = 0;
			for (int kInd = 0; kInd < k; ++kInd)
			{
				const float d = searchPoints[kInd + mInd * k] - referencePoints[kInd + nInd * k];
				dis += d * d;
			}
			if (dis_s[threadIdx.x] > dis)
			{
				dis_s[threadIdx.x] = dis;
				ind_s[threadIdx.x] = nInd;
			}
		}
		__syncthreads();
		for (int offset = BLOCK_DIM_X >> 1; offset > 0; offset >>= 1)
		{
			if (threadIdx.x < offset)
				if (dis_s[threadIdx.x] > dis_s[threadIdx.x ^ offset])
				{
					dis_s[threadIdx.x] = dis_s[threadIdx.x ^ offset];
					ind_s[threadIdx.x] = ind_s[threadIdx.x ^ offset];
				}
			__syncthreads();
		}
		if (threadIdx.x == 0)
			result[ans_id] = ind_s[0];
	}
	static void cudaCallback(
		int k,
		int m,
		int n,
		float *searchPoints,
		float *referencePoints,
		int **results)
	{
		thrust::device_vector<int> results_d(m);
		{
			thrust::device_vector<float>
				s_d(searchPoints, searchPoints + k * m),
				r_d(referencePoints, referencePoints + k * n);
			const int BLOCK_DIM_X = 1024;
			//WuKTimer t1;
			cudaCallbackKernel<
				BLOCK_DIM_X><<<
				dim3(results_d.size() / m, m),
				BLOCK_DIM_X>>>(
				k,
				m,
				n,
				results_d.size(),
				thrust::raw_pointer_cast(s_d.data()),
				thrust::raw_pointer_cast(r_d.data()),
				thrust::raw_pointer_cast(results_d.data()));
		}
		thrust::copy(
			results_d.begin(),
			results_d.begin() + m,
			*results = (int *)malloc(sizeof(int) * m));
	}
}; // namespace v3
namespace v4
{
	template <int BLOCK_DIM_X>
	static __global__ void
	cudaCallbackKernel(
		const int k,
		const int m,
		const int n,
		const int result_size,
		const float *__restrict__ referencePoints,
		int *__restrict__ result)
	{
		const int ans_id = blockIdx.x + blockIdx.y * gridDim.x;
		if (ans_id >= result_size)
			return;
		__shared__ float dis_s[BLOCK_DIM_X];
		__shared__ int ind_s[BLOCK_DIM_X];
		dis_s[threadIdx.x] = INFINITY;
		for (int mInd = blockIdx.y, nInd = threadIdx.x + blockIdx.x * BLOCK_DIM_X;
			 nInd < n;
			 nInd += gridDim.x * BLOCK_DIM_X)
		{
			float dis = 0;
			for (int kInd = 0; kInd < k; ++kInd)
			{
				const float d = const_mem[kInd + mInd * k] - referencePoints[kInd + nInd * k];
				dis += d * d;
			}
			if (dis_s[threadIdx.x] > dis)
			{
				dis_s[threadIdx.x] = dis;
				ind_s[threadIdx.x] = nInd;
			}
		}
		__syncthreads();
		for (int offset = BLOCK_DIM_X >> 1; offset > 0; offset >>= 1)
		{
			if (threadIdx.x < offset)
				if (dis_s[threadIdx.x] > dis_s[threadIdx.x ^ offset])
				{
					dis_s[threadIdx.x] = dis_s[threadIdx.x ^ offset];
					ind_s[threadIdx.x] = ind_s[threadIdx.x ^ offset];
				}
			__syncthreads();
		}
		if (threadIdx.x == 0)
			result[ans_id] = ind_s[0];
	}
	static void cudaCallback(
		int k,
		int m,
		int n,
		float *searchPoints,
		float *referencePoints,
		int **results)
	{
		assert(k * m <= (64 << 10) / sizeof(float));
		CHECK(hipMemcpyToSymbol(HIP_SYMBOL(const_mem), searchPoints, sizeof(float) * k * m));
		thrust::device_vector<int> results_d(m);
		{
			thrust::device_vector<float>
				r_d(referencePoints, referencePoints + k * n);
			const int BLOCK_DIM_X = 1024;
			//WuKTimer t1;
			cudaCallbackKernel<
				BLOCK_DIM_X><<<
				dim3(results_d.size() / m, m),
				BLOCK_DIM_X>>>(
				k,
				m,
				n,
				results_d.size(),
				thrust::raw_pointer_cast(r_d.data()),
				thrust::raw_pointer_cast(results_d.data()));
		}
		thrust::copy(
			results_d.begin(),
			results_d.begin() + m,
			*results = (int *)malloc(sizeof(int) * m));
	}
}; // namespace v4
namespace v5
{
	template <int BLOCK_DIM_X>
	static __global__ void
	cudaCallbackKernel(
		const int k,
		const int m,
		const int n,
		const int result_size,
		hipTextureObject_t texObj, //使用纹理对象
		int *__restrict__ result)
	{
		const int ans_id = blockIdx.x + blockIdx.y * gridDim.x;
		if (ans_id >= result_size)
			return;
		__shared__ float dis_s[BLOCK_DIM_X];
		__shared__ int ind_s[BLOCK_DIM_X];
		dis_s[threadIdx.x] = INFINITY;
		for (int mInd = blockIdx.y, nInd = threadIdx.x + blockIdx.x * BLOCK_DIM_X;
			 nInd < n;
			 nInd += gridDim.x * BLOCK_DIM_X)
		{
			float dis = 0;
			for (int kInd = 0; kInd < k; ++kInd)
			{
				const float d = const_mem[kInd + mInd * k] - tex2D<float>(texObj, kInd, nInd);
				dis += d * d;
			}
			if (dis_s[threadIdx.x] > dis)
			{
				dis_s[threadIdx.x] = dis;
				ind_s[threadIdx.x] = nInd;
			}
		}
		__syncthreads();
		for (int offset = BLOCK_DIM_X >> 1; offset > 0; offset >>= 1)
		{
			if (threadIdx.x < offset)
				if (dis_s[threadIdx.x] > dis_s[threadIdx.x ^ offset])
				{
					dis_s[threadIdx.x] = dis_s[threadIdx.x ^ offset];
					ind_s[threadIdx.x] = ind_s[threadIdx.x ^ offset];
				}
			__syncthreads();
		}
		if (threadIdx.x == 0)
			result[ans_id] = ind_s[0];
	}
	static void cudaCallback(
		int k,
		int m,
		int n,
		float *searchPoints,
		float *referencePoints,
		int **results)
	{
		assert(k * m <= (64 << 10) / sizeof(float));
		CHECK(hipMemcpyToSymbol(HIP_SYMBOL(const_mem), searchPoints, sizeof(float) * k * m));
		hipArray *cuArray;
		hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
		CHECK(hipMallocArray(&cuArray, &channelDesc, k, n));
		CHECK(hipMemcpy2DToArray(cuArray, 0, 0, referencePoints, sizeof(float) * k, sizeof(float) * k, n, hipMemcpyHostToDevice));

		// 绑定纹理到hipArray上
		struct hipResourceDesc resDesc;
		memset(&resDesc, 0, sizeof(resDesc));
		resDesc.resType = hipResourceTypeArray;
		resDesc.res.array.array = cuArray;

		// 设置纹理为只读
		struct hipTextureDesc texDesc;
		memset(&texDesc, 0, sizeof(texDesc));
		texDesc.readMode = hipReadModeElementType;

		// 创建纹理对象
		hipTextureObject_t texObj = 0;
		CHECK(hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL));

		thrust::device_vector<int> results_d(m);
		{
			const int BLOCK_DIM_X = 1024;
			//WuKTimer t1;
			cudaCallbackKernel<
				BLOCK_DIM_X><<<
				dim3(results_d.size() / m, m),
				BLOCK_DIM_X>>>(
				k,
				m,
				n,
				results_d.size(),
				texObj,
				thrust::raw_pointer_cast(results_d.data()));
		}
		thrust::copy(
			results_d.begin(),
			results_d.begin() + m,
			*results = (int *)malloc(sizeof(int) * m));
	}
}; // namespace v5
struct WarmUP
{
	WarmUP(int k, int m, int n)
	{
		void (*cudaCallback[])(int, int, int, float *, float *, int **) = {
			v0::cudaCallback,
			v1::cudaCallback,
			v3::cudaCallback,
			v4::cudaCallback,
			v5::cudaCallback}; //由于多卡版本是调用单卡版本实现的，因此无需热身
		float *searchPoints = (float *)malloc(sizeof(float) * k * m);
		float *referencePoints = (float *)malloc(sizeof(float) * k * n);

#pragma omp parallel
		{
			unsigned seed = omp_get_thread_num(); //每个线程使用不同的随机数种子
#pragma omp for
			for (int i = 0; i < k * m; ++i)
				searchPoints[i] = rand_r(&seed) / double(RAND_MAX); //使用线程安全的随机数函数
#pragma omp for
			for (int i = 0; i < k * n; ++i)
				referencePoints[i] = rand_r(&seed) / double(RAND_MAX);
		}

		for (int i = 0; i < sizeof(cudaCallback) / sizeof(cudaCallback[0]); ++i)
		{
			int num_gpus = 0;
			CHECK(hipGetDeviceCount(&num_gpus));
#pragma omp parallel num_threads(num_gpus) //对于每张显卡都要优化
			{
				int *result;
				int thread_num = omp_get_thread_num();
				CHECK(hipSetDevice(thread_num));
				cudaCallback[i](k, m, n, searchPoints, referencePoints, &result);
				free(result);
			}
		}
		free(searchPoints);
		free(referencePoints);
	}
};
struct BenchMark
{
	BenchMark(int k, int m, int n)
	{
		void (*cudaCallback[])(int, int, int, float *, float *, int **) = {
			v0::cudaCallback,
			v1::cudaCallback,
			v2::cudaCallback,
			v3::cudaCallback,
			v4::cudaCallback,
			v5::cudaCallback}; //由于多卡版本是调用单卡版本实现的，因此无需热身
		float *searchPoints = (float *)malloc(sizeof(float) * k * m);
		float *referencePoints = (float *)malloc(sizeof(float) * k * n);

#pragma omp parallel
		{
			unsigned seed = omp_get_thread_num(); //每个线程使用不同的随机数种子
#pragma omp for
			for (int i = 0; i < k * m; ++i)
				searchPoints[i] = rand_r(&seed) / double(RAND_MAX); //使用线程安全的随机数函数
#pragma omp for
			for (int i = 0; i < k * n; ++i)
				referencePoints[i] = rand_r(&seed) / double(RAND_MAX);
		}
		printf("\n\nStart benchmark with (k, m, n) = %d, %d, %d:\n\n", k, m, n); //开始benchnmark
		for (int i = 0; i < sizeof(cudaCallback) / sizeof(cudaCallback[0]); ++i)
		{
			int *result;
			printf("Version %d: ", i);
			{
				WuKTimer t1;
				cudaCallback[i](k, m, n, searchPoints, referencePoints, &result);
			}
			free(result);
			fflush(stdout);
		}
		printf("\nFinish benchmark with (k, m, n) = (%d, %d, %d).\n\n", k, m, n);
		free(searchPoints);
		free(referencePoints);
	}
};
static WarmUP warm_up(1, 1, 1);
static BenchMark
	benchmark1(16384, 1, 65536),
	benchmark1024(16, 1024, 65536);
void cudaCallback(
	int k,
	int m,
	int n,
	float *searchPoints,
	float *referencePoints,
	int **results)
{
	v5::cudaCallback(
		k,
		m,
		n,
		searchPoints,
		referencePoints,
		results);
}