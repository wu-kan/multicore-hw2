#include "hip/hip_runtime.h"
#include "core.h"

struct WuKTimer
{
	hipEvent_t beg, end;
	WuKTimer()
	{
		hipEventCreate(&beg);
		hipEventCreate(&end);
		hipEventRecord(beg);
	}
	~WuKTimer()
	{
		hipEventRecord(end);
		hipEventSynchronize(beg);
		hipEventSynchronize(end);
		float elapsed_time;
		hipEventElapsedTime(
			&elapsed_time,
			beg,
			end);
		printf("%f\n", elapsed_time);
	}
};
namespace v0
{
	static void cudaCallback(
		int k,
		int m,
		int n,
		float *searchPoints,
		float *referencePoints,
		int **results)
	{

		int *tmp = (int *)malloc(sizeof(int) * m);
		int minIndex;
		float minSquareSum, diff, squareSum;

		// Iterate over all search points
		for (int mInd = 0; mInd < m; mInd++)
		{
			minSquareSum = -1;
			// Iterate over all reference points
			for (int nInd = 0; nInd < n; nInd++)
			{
				squareSum = 0;
				for (int kInd = 0; kInd < k; kInd++)
				{
					diff = searchPoints[k * mInd + kInd] - referencePoints[k * nInd + kInd];
					squareSum += (diff * diff);
				}
				if (minSquareSum < 0 || squareSum < minSquareSum)
				{
					minSquareSum = squareSum;
					minIndex = nInd;
				}
			}
			tmp[mInd] = minIndex;
		}

		*results = tmp;
		// Note that you don't have to free searchPoints, referencePoints, and
		// *results by yourself
	}
} // namespace v0
namespace v1
{
	static __global__ void
	get_dis_kernel(
		const int k,
		const int m,
		const int n,
		const float *__restrict__ searchPoints,
		const float *__restrict__ referencePoints,
		float *__restrict__ dis)
	{
		const int
			nInd = threadIdx.x + blockIdx.x * blockDim.x,
			mInd = threadIdx.y + blockIdx.y * blockDim.y;
		if (nInd < n && mInd < m)
		{
			float squareSum = 0;
			for (int kInd = 0; kInd < k; ++kInd)
			{
				const float diff = searchPoints[kInd + mInd * k] - referencePoints[kInd + nInd * k];
				squareSum += diff * diff;
			}
			dis[nInd + mInd * n] = squareSum;
		}
	}
	static void cudaCallback(
		int k,
		int m,
		int n,
		float *searchPoints,
		float *referencePoints,
		int **results)
	{
		thrust::device_vector<float> dis_d(m * n);
		{
			thrust::device_vector<float>
				s_d(searchPoints, searchPoints + k * m),
				r_d(referencePoints, referencePoints + k * n);
			const int BLOCK_DIM_X = 32, BLOCK_DIM_Y = 32;
			//WuKTimer t1;
			get_dis_kernel<<<
				dim3(divup(n, BLOCK_DIM_X), divup(m, BLOCK_DIM_Y)),
				dim3(BLOCK_DIM_X, BLOCK_DIM_Y)>>>(
				k,
				m,
				n,
				thrust::raw_pointer_cast(s_d.data()),
				thrust::raw_pointer_cast(r_d.data()),
				thrust::raw_pointer_cast(dis_d.data()));
		}
		*results = (int *)malloc(sizeof(int) * m);
		{
			//WuKTimer t2;
			for (int i = 0; i < m; ++i)
				(*results)[i] = thrust::min_element(dis_d.begin() + n * i, dis_d.begin() + n * i + n) - dis_d.begin() - n * i;
		}
	}
}; // namespace v1
namespace v2
{
	__global__ void static get_dis_kernel(
		const int k,
		const int m,
		const int n,
		const float *__restrict__ searchPoints,
		const float *__restrict__ referencePoints,
		float *__restrict__ dis)
	{
		const int
			nInd = threadIdx.x + blockIdx.x * blockDim.x,
			mInd = threadIdx.y + blockIdx.y * blockDim.y;
		if (nInd < n && mInd < m)
		{
			float squareSum = 0;
			for (int kInd = 0; kInd < k; ++kInd)
			{
				const float diff = searchPoints[kInd + mInd * k] - referencePoints[kInd + nInd * k];
				squareSum += diff * diff;
			}
			dis[nInd + mInd * n] = squareSum;
		}
	}
	template <int BLOCK_DIM_X>
	static __global__ void
	get_min_kernel(
		const int result_size,
		const int m,
		const int n,
		const float *__restrict__ dis,
		int *__restrict__ result)
	{
		const int ans_id = blockIdx.x * gridDim.y + blockIdx.y;
		if (ans_id >= result_size)
			return;
		__shared__ float dis_s[BLOCK_DIM_X];
		__shared__ int ind_s[BLOCK_DIM_X];
		dis_s[threadIdx.x] = INFINITY;
		for (int nInd = threadIdx.x + blockIdx.x * BLOCK_DIM_X;
			 nInd < n;
			 nInd += gridDim.x * BLOCK_DIM_X)
		{
			const float squareSum = dis[nInd + blockIdx.y * n];
			if (dis_s[threadIdx.x] > squareSum)
			{
				dis_s[threadIdx.x] = squareSum;
				ind_s[threadIdx.x] = nInd;
			}
		}
		__syncthreads();
		for (int offset = BLOCK_DIM_X >> 1; offset > 0; offset >>= 1)
		{
			if (threadIdx.x < offset)
				if (dis_s[threadIdx.x] > dis_s[threadIdx.x ^ offset])
				{
					dis_s[threadIdx.x] = dis_s[threadIdx.x ^ offset];
					ind_s[threadIdx.x] = ind_s[threadIdx.x ^ offset];
				}
			__syncthreads();
		}
		if (threadIdx.x == 0)
			result[ans_id] = ind_s[0];
	}
	static void cudaCallback(
		int k,
		int m,
		int n,
		float *searchPoints,
		float *referencePoints,
		int **results)
	{
		thrust::device_vector<float> dis_d(m * n);
		{
			thrust::device_vector<float>
				s_d(searchPoints, searchPoints + k * m),
				r_d(referencePoints, referencePoints + k * n);
			const int BLOCK_DIM_X = 32, BLOCK_DIM_Y = 32;
			//WuKTimer t1;
			get_dis_kernel<<<
				dim3(divup(n, BLOCK_DIM_X), divup(m, BLOCK_DIM_Y)),
				dim3(BLOCK_DIM_X, BLOCK_DIM_Y)>>>(
				k,
				m,
				n,
				thrust::raw_pointer_cast(s_d.data()),
				thrust::raw_pointer_cast(r_d.data()),
				thrust::raw_pointer_cast(dis_d.data()));
		}
		thrust::device_vector<int> results_d(m);
		{
			const int BLOCK_DIM_X = 1024;
			//WuKTimer t2;
			get_min_kernel<
				BLOCK_DIM_X><<<
				dim3(results_d.size() / m, m),
				BLOCK_DIM_X>>>(
				results_d.size(),
				m,
				n,
				thrust::raw_pointer_cast(dis_d.data()),
				thrust::raw_pointer_cast(results_d.data()));
		}
		thrust::copy(
			results_d.begin(),
			results_d.end(),
			*results = (int *)malloc(sizeof(int) * m));
	}
}; // namespace v2
namespace v3
{
	template <int BLOCK_DIM_X>
	static __global__ void
	cudaCallbackKernel(
		const int k,
		const int m,
		const int n,
		const int result_size,
		const float *__restrict__ searchPoints,
		const float *__restrict__ referencePoints,
		int *__restrict__ result)
	{
		const int ans_id = blockIdx.x * gridDim.y + blockIdx.y;
		if (ans_id >= result_size)
			return;
		__shared__ float dis_s[BLOCK_DIM_X];
		__shared__ int ind_s[BLOCK_DIM_X];
		dis_s[threadIdx.x] = INFINITY;
		for (int mInd = blockIdx.y, nInd = threadIdx.x + blockIdx.x * BLOCK_DIM_X;
			 nInd < n;
			 nInd += gridDim.x * BLOCK_DIM_X)
		{
			float squareSum = 0;
			for (int kInd = 0; kInd < k; ++kInd)
			{
				const float diff = searchPoints[kInd + mInd * k] - referencePoints[kInd + nInd * k];
				squareSum += diff * diff;
			}
			if (dis_s[threadIdx.x] > squareSum)
			{
				dis_s[threadIdx.x] = squareSum;
				ind_s[threadIdx.x] = nInd;
			}
		}
		__syncthreads();
		for (int offset = BLOCK_DIM_X >> 1; offset > 0; offset >>= 1)
		{
			if (threadIdx.x < offset)
				if (dis_s[threadIdx.x] > dis_s[threadIdx.x ^ offset])
				{
					dis_s[threadIdx.x] = dis_s[threadIdx.x ^ offset];
					ind_s[threadIdx.x] = ind_s[threadIdx.x ^ offset];
				}
			__syncthreads();
		}
		if (threadIdx.x == 0)
			result[ans_id] = ind_s[0];
	}
	static void cudaCallback(
		int k,
		int m,
		int n,
		float *searchPoints,
		float *referencePoints,
		int **results)
	{
		thrust::device_vector<int> results_d(m);
		{
			thrust::device_vector<float>
				s_d(searchPoints, searchPoints + k * m),
				r_d(referencePoints, referencePoints + k * n);
			const int BLOCK_DIM_X = 1024;
			//WuKTimer t1;
			cudaCallbackKernel<
				BLOCK_DIM_X><<<
				dim3(results_d.size() / m, m),
				BLOCK_DIM_X>>>(
				k,
				m,
				n,
				results_d.size(),
				thrust::raw_pointer_cast(s_d.data()),
				thrust::raw_pointer_cast(r_d.data()),
				thrust::raw_pointer_cast(results_d.data()));
		}
		thrust::copy(
			results_d.begin(),
			results_d.end(),
			*results = (int *)malloc(sizeof(int) * m));
	}
}; // namespace v3
namespace v4
{
	static __global__ void
	mat_inv_kernel(
		const int k,
		const int n,
		const float *__restrict__ input,
		float *__restrict__ output)
	{
		const int
			nInd = threadIdx.x + blockIdx.x * blockDim.x,
			kInd = threadIdx.y + blockIdx.y * blockDim.y;
		if (nInd < n && kInd < k)
		{
			const float a = input[nInd * k + kInd];
			output[nInd + kInd * n] = a;
		}
	}
	template <int BLOCK_DIM_X>
	static __global__ void
	cudaCallbackKernel(
		const int k,
		const int m,
		const int n,
		const int result_size,
		const float *__restrict__ searchPoints,
		const float *__restrict__ referencePoints,
		int *__restrict__ result)
	{
		const int ans_id = blockIdx.x * gridDim.y + blockIdx.y;
		if (ans_id >= result_size)
			return;
		__shared__ float dis_s[BLOCK_DIM_X];
		__shared__ int ind_s[BLOCK_DIM_X];
		dis_s[threadIdx.x] = INFINITY;
		for (int mInd = blockIdx.y, nInd = threadIdx.x + blockIdx.x * BLOCK_DIM_X;
			 nInd < n;
			 nInd += gridDim.x * BLOCK_DIM_X)
		{
			float squareSum = 0;
			for (int kInd = 0; kInd < k; ++kInd)
			{
				const float diff = searchPoints[kInd + mInd * k] - referencePoints[kInd * n + nInd];
				squareSum += diff * diff;
			}
			if (dis_s[threadIdx.x] > squareSum)
			{
				dis_s[threadIdx.x] = squareSum;
				ind_s[threadIdx.x] = nInd;
			}
		}
		__syncthreads();
		for (int offset = BLOCK_DIM_X >> 1; offset > 0; offset >>= 1)
		{
			if (threadIdx.x < offset)
				if (dis_s[threadIdx.x] > dis_s[threadIdx.x ^ offset])
				{
					dis_s[threadIdx.x] = dis_s[threadIdx.x ^ offset];
					ind_s[threadIdx.x] = ind_s[threadIdx.x ^ offset];
				}
			__syncthreads();
		}
		if (threadIdx.x == 0)
			result[ans_id] = ind_s[0];
	}
	static void cudaCallback(
		int k,
		int m,
		int n,
		float *searchPoints,
		float *referencePoints,
		int **results)
	{
		thrust::device_vector<int> results_d(m);
		thrust::device_vector<float>
			s_d(searchPoints, searchPoints + k * m),
			r_d(k * n);
		{
			thrust::device_vector<float>
				rr_d(referencePoints, referencePoints + k * n);
			const int BLOCK_DIM_X = 32, BLOCK_DIM_Y = 32;
			//WuKTimer t1;
			mat_inv_kernel<<<
				dim3(divup(n, BLOCK_DIM_X), divup(k, BLOCK_DIM_Y)),
				dim3(BLOCK_DIM_X, BLOCK_DIM_Y)>>>(
				k,
				n,
				thrust::raw_pointer_cast(rr_d.data()),
				thrust::raw_pointer_cast(r_d.data()));
		}
		{
			const int BLOCK_DIM_X = 1024;
			//WuKTimer t1;
			cudaCallbackKernel<
				BLOCK_DIM_X><<<
				dim3(results_d.size() / m, m),
				BLOCK_DIM_X>>>(
				k,
				m,
				n,
				results_d.size(),
				thrust::raw_pointer_cast(s_d.data()),
				thrust::raw_pointer_cast(r_d.data()),
				thrust::raw_pointer_cast(results_d.data()));
		}
		thrust::copy(
			results_d.begin(),
			results_d.end(),
			*results = (int *)malloc(sizeof(int) * m));
	}
}; // namespace v4
namespace v5
{
	template <int BLOCK_DIM_X>
	static __global__ void
	cudaCallbackKernel(
		const int k,
		const int m,
		const int n,
		const int result_size,
		const float *__restrict__ searchPoints,
		hipTextureObject_t texObj, //使用纹理对象
		int *__restrict__ result)
	{
		const int ans_id = blockIdx.x * gridDim.y + blockIdx.y;
		if (ans_id >= result_size)
			return;
		__shared__ float dis_s[BLOCK_DIM_X];
		__shared__ int ind_s[BLOCK_DIM_X];
		dis_s[threadIdx.x] = INFINITY;
		for (int mInd = blockIdx.y, nInd = threadIdx.x + blockIdx.x * BLOCK_DIM_X;
			 nInd < n;
			 nInd += gridDim.x * BLOCK_DIM_X)
		{
			float squareSum = 0;
			for (int kInd = 0; kInd < k; ++kInd)
			{
				const float diff = searchPoints[kInd + mInd * k] - tex2D<float>(texObj, kInd, nInd);
				squareSum += diff * diff;
			}
			if (dis_s[threadIdx.x] > squareSum)
			{
				dis_s[threadIdx.x] = squareSum;
				ind_s[threadIdx.x] = nInd;
			}
		}
		__syncthreads();
		for (int offset = BLOCK_DIM_X >> 1; offset > 0; offset >>= 1)
		{
			if (threadIdx.x < offset)
				if (dis_s[threadIdx.x] > dis_s[threadIdx.x ^ offset])
				{
					dis_s[threadIdx.x] = dis_s[threadIdx.x ^ offset];
					ind_s[threadIdx.x] = ind_s[threadIdx.x ^ offset];
				}
			__syncthreads();
		}
		if (threadIdx.x == 0)
			result[ans_id] = ind_s[0];
	}
	static void cudaCallback(
		int k,
		int m,
		int n,
		float *searchPoints,
		float *referencePoints,
		int **results)
	{
		if (n > 65536)
		{
			v4::cudaCallback(k, m, n, searchPoints, referencePoints, results);
			return;
		}
		hipArray *cuArray;
		hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
		CHECK(hipMallocArray(&cuArray, &channelDesc, k, n));
		CHECK(hipMemcpy2DToArray(cuArray, 0, 0, referencePoints, sizeof(float) * k, sizeof(float) * k, n, hipMemcpyHostToDevice));

		// 绑定纹理到hipArray上
		struct hipResourceDesc resDesc;
		memset(&resDesc, 0, sizeof(resDesc));
		resDesc.resType = hipResourceTypeArray;
		resDesc.res.array.array = cuArray;

		// 设置纹理为只读
		struct hipTextureDesc texDesc;
		memset(&texDesc, 0, sizeof(texDesc));
		texDesc.readMode = hipReadModeElementType;

		// 创建纹理对象
		hipTextureObject_t texObj = 0;
		CHECK(hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL));

		thrust::device_vector<int> results_d(m);
		thrust::device_vector<float>
			s_d(searchPoints, searchPoints + k * m);
		{
			const int BLOCK_DIM_X = 1024;
			//WuKTimer t1;
			cudaCallbackKernel<
				BLOCK_DIM_X><<<
				dim3(results_d.size() / m, m),
				BLOCK_DIM_X>>>(
				k,
				m,
				n,
				results_d.size(),
				thrust::raw_pointer_cast(s_d.data()),
				texObj,
				thrust::raw_pointer_cast(results_d.data()));
		}
		thrust::copy(
			results_d.begin(),
			results_d.end(),
			*results = (int *)malloc(sizeof(int) * m));
	}
}; // namespace v5
namespace v6
{
	static __constant__ float const_mem[(64 << 10) / sizeof(float)];
	static __global__ void
	mat_inv_kernel(
		const int k,
		const int n,
		const float *__restrict__ input,
		float *__restrict__ output)
	{
		const int
			nInd = threadIdx.x + blockIdx.x * blockDim.x,
			kInd = threadIdx.y + blockIdx.y * blockDim.y;
		if (nInd < n && kInd < k)
		{
			const float a = input[nInd * k + kInd];
			output[nInd + kInd * n] = a;
		}
	}
	template <int BLOCK_DIM_X>
	static __global__ void
	cudaCallbackKernel(
		const int k,
		const int m,
		const int n,
		const int result_size,
		const float *__restrict__ referencePoints,
		int *__restrict__ result)
	{
		const int ans_id = blockIdx.x * gridDim.y + blockIdx.y;
		if (ans_id >= result_size)
			return;
		__shared__ float dis_s[BLOCK_DIM_X];
		__shared__ int ind_s[BLOCK_DIM_X];
		dis_s[threadIdx.x] = INFINITY;
		for (int mInd = blockIdx.y, nInd = threadIdx.x + blockIdx.x * BLOCK_DIM_X;
			 nInd < n;
			 nInd += gridDim.x * BLOCK_DIM_X)
		{
			float squareSum = 0;
			for (int kInd = 0; kInd < k; ++kInd)
			{
				const float diff = const_mem[kInd + mInd * k] - referencePoints[kInd * n + nInd];
				squareSum += diff * diff;
			}
			if (dis_s[threadIdx.x] > squareSum)
			{
				dis_s[threadIdx.x] = squareSum;
				ind_s[threadIdx.x] = nInd;
			}
		}
		__syncthreads();
		for (int offset = BLOCK_DIM_X >> 1; offset > 0; offset >>= 1)
		{
			if (threadIdx.x < offset)
				if (dis_s[threadIdx.x] > dis_s[threadIdx.x ^ offset])
				{
					dis_s[threadIdx.x] = dis_s[threadIdx.x ^ offset];
					ind_s[threadIdx.x] = ind_s[threadIdx.x ^ offset];
				}
			__syncthreads();
		}
		if (threadIdx.x == 0)
			result[ans_id] = ind_s[0];
	}
	static void cudaCallback(
		int k,
		int m,
		int n,
		float *searchPoints,
		float *referencePoints,
		int **results)
	{
		if (k * m > (64 << 10) / sizeof(float))
		{
			v4::cudaCallback(k, m, n, searchPoints, referencePoints, results);
			return;
		}
		CHECK(hipMemcpyToSymbol(HIP_SYMBOL(const_mem), searchPoints, sizeof(float) * k * m));
		thrust::device_vector<int> results_d(m);
		thrust::device_vector<float> r_d(k * n);
		{
			thrust::device_vector<float>
				rr_d(referencePoints, referencePoints + k * n);
			const int BLOCK_DIM_X = 32, BLOCK_DIM_Y = 32;
			//WuKTimer t1;
			mat_inv_kernel<<<
				dim3(divup(n, BLOCK_DIM_X), divup(k, BLOCK_DIM_Y)),
				dim3(BLOCK_DIM_X, BLOCK_DIM_Y)>>>(
				k,
				n,
				thrust::raw_pointer_cast(rr_d.data()),
				thrust::raw_pointer_cast(r_d.data()));
		}
		{
			const int BLOCK_DIM_X = 1024;
			//WuKTimer t2;
			cudaCallbackKernel<
				BLOCK_DIM_X><<<
				dim3(results_d.size() / m, m),
				BLOCK_DIM_X>>>(
				k,
				m,
				n,
				results_d.size(),
				thrust::raw_pointer_cast(r_d.data()),
				thrust::raw_pointer_cast(results_d.data()));
		}
		thrust::copy(
			results_d.begin(),
			results_d.end(),
			*results = (int *)malloc(sizeof(int) * m));
	}
}; // namespace v6
namespace v7
{
	static __global__ void
	mat_inv_kernel(
		const int k,
		const int n,
		const float *__restrict__ input,
		float *__restrict__ output)
	{
		const int
			nInd = threadIdx.x + blockIdx.x * blockDim.x,
			kInd = threadIdx.y + blockIdx.y * blockDim.y;
		if (nInd < n && kInd < k)
		{
			const float a = input[nInd * k + kInd];
			output[nInd + kInd * n] = a;
		}
	}
	template <int BLOCK_DIM_X>
	static __global__ void
	cudaCallbackKernel(
		const int k,
		const int m,
		const int n,
		const int result_size,
		const float *__restrict__ searchPoints,
		const float *__restrict__ referencePoints,
		int *__restrict__ result)
	{
		const int ans_id = blockIdx.x * gridDim.y + blockIdx.y;
		if (ans_id >= result_size)
			return;
		__shared__ float dis_s[BLOCK_DIM_X];
		__shared__ int ind_s[BLOCK_DIM_X];
		dis_s[threadIdx.x] = INFINITY;
		ind_s[threadIdx.x] = 0;
		for (int mInd = blockIdx.y, nInd = threadIdx.x + blockIdx.x * BLOCK_DIM_X;
			 nInd < n;
			 nInd += gridDim.x * BLOCK_DIM_X)
		{
			float squareSum = 0;
			for (int kInd = 0; kInd < k; ++kInd)
			{
				const float diff = searchPoints[kInd + mInd * k] - referencePoints[kInd * n + nInd];
				squareSum += diff * diff;
			}
			if (dis_s[threadIdx.x] > squareSum)
			{
				dis_s[threadIdx.x] = squareSum;
				ind_s[threadIdx.x] = nInd;
			}
		}
		__syncthreads();
		for (int offset = BLOCK_DIM_X >> 1; offset > 0; offset >>= 1)
		{
			if (threadIdx.x < offset)
				if (dis_s[threadIdx.x] > dis_s[threadIdx.x ^ offset])
				{
					dis_s[threadIdx.x] = dis_s[threadIdx.x ^ offset];
					ind_s[threadIdx.x] = ind_s[threadIdx.x ^ offset];
				}
			__syncthreads();
		}
		if (threadIdx.x == 0)
			result[ans_id] = ind_s[0];
	}
	static void cudaCallback(
		int k,
		int m,
		int n,
		float *searchPoints,
		float *referencePoints,
		int **results)
	{
		thrust::device_vector<float>
			s_d(searchPoints, searchPoints + k * m),
			r_d(k * n);
		{
			thrust::device_vector<float>
				rr_d(referencePoints, referencePoints + k * n);
			const int BLOCK_DIM_X = 32, BLOCK_DIM_Y = 32;
			//WuKTimer t1;
			mat_inv_kernel<<<
				dim3(divup(n, BLOCK_DIM_X), divup(k, BLOCK_DIM_Y)),
				dim3(BLOCK_DIM_X, BLOCK_DIM_Y)>>>(
				k,
				n,
				thrust::raw_pointer_cast(rr_d.data()),
				thrust::raw_pointer_cast(r_d.data()));
		}
		const int BLOCK_DIM_X = 1024;
		int numBlocks;
		CHECK(hipOccupancyMaxActiveBlocksPerMultiprocessor(
			&numBlocks,
			cudaCallbackKernel<BLOCK_DIM_X>,
			BLOCK_DIM_X,
			0));
		thrust::device_vector<int> results_d(m * divup(numBlocks, m));
		{
			//WuKTimer t2;
			cudaCallbackKernel<
				BLOCK_DIM_X><<<
				dim3(results_d.size() / m, m),
				BLOCK_DIM_X>>>(
				k,
				m,
				n,
				results_d.size(),
				thrust::raw_pointer_cast(s_d.data()),
				thrust::raw_pointer_cast(r_d.data()),
				thrust::raw_pointer_cast(results_d.data()));
		}
		*results = (int *)malloc(sizeof(int) * m);
		if (results_d.size() == m)
		{
			thrust::copy(
				results_d.begin(),
				results_d.end(),
				*results);
			return;
		}
		thrust::host_vector<int> results_tmp(results_d);
#pragma omp parallel for
		for (int mInd = 0; mInd < m; ++mInd)
		{
			float minSquareSum = INFINITY;
			int minIndex = 0;
			// Iterate over all reference points
			for (int i = 0; i < results_tmp.size(); i += m)
			{
				const int nInd = results_tmp[i];
				float squareSum = 0;
				for (int kInd = 0; kInd < k; ++kInd)
				{
					const float diff = searchPoints[k * mInd + kInd] - referencePoints[k * nInd + kInd];
					squareSum += diff * diff;
				}
				if (minSquareSum > squareSum)
				{
					minSquareSum = squareSum;
					minIndex = nInd;
				}
			}
			(*results)[mInd] = minIndex;
		}
	}
}; // namespace v7
namespace v8
{
	static __global__ void
	mat_inv_kernel(
		const int k,
		const int n,
		const float *__restrict__ input,
		float *__restrict__ output)
	{
		const int
			nInd = threadIdx.x + blockIdx.x * blockDim.x,
			kInd = threadIdx.y + blockIdx.y * blockDim.y;
		if (nInd < n && kInd < k)
		{
			const float a = input[nInd * k + kInd];
			output[nInd + kInd * n] = a;
		}
	}
	template <int BLOCK_DIM_X>
	static __global__ void
	cudaCallbackKernel(
		const int k,
		const int m,
		const int n,
		const int result_size,
		const float *__restrict__ searchPoints,
		const float *__restrict__ referencePoints,
		int *__restrict__ result)
	{
		const int ans_id = blockIdx.x * gridDim.y + blockIdx.y;
		if (ans_id >= result_size)
			return;
		__shared__ float dis_s[BLOCK_DIM_X];
		__shared__ int ind_s[BLOCK_DIM_X];
		dis_s[threadIdx.x] = INFINITY;
		ind_s[threadIdx.x] = 0;
		for (int mInd = blockIdx.y, nInd = threadIdx.x + blockIdx.x * BLOCK_DIM_X;
			 nInd < n;
			 nInd += gridDim.x * BLOCK_DIM_X)
		{
			float squareSum = 0;
			for (int kInd = 0; kInd < k; ++kInd)
			{
				const float diff = searchPoints[kInd + mInd * k] - referencePoints[kInd * n + nInd];
				squareSum += diff * diff;
			}
			if (dis_s[threadIdx.x] > squareSum)
			{
				dis_s[threadIdx.x] = squareSum;
				ind_s[threadIdx.x] = nInd;
			}
		}
		__syncthreads();
		for (int offset = BLOCK_DIM_X >> 1; offset > 0; offset >>= 1)
		{
			if (threadIdx.x < offset)
				if (dis_s[threadIdx.x] > dis_s[threadIdx.x ^ offset])
				{
					dis_s[threadIdx.x] = dis_s[threadIdx.x ^ offset];
					ind_s[threadIdx.x] = ind_s[threadIdx.x ^ offset];
				}
			__syncthreads();
		}
		if (threadIdx.x == 0)
			result[ans_id] = ind_s[0];
	}
	static void cudaCallback(
		int k,
		int m,
		int n,
		float *searchPoints,
		float *referencePoints,
		int **results)
	{
		thrust::host_vector<int> results_tmp;
		int num_gpus = 0;
		CHECK(hipGetDeviceCount(&num_gpus));
		if (num_gpus > n)
			num_gpus = n;
		if (num_gpus < 1)
			return v0::cudaCallback(k, m, n, searchPoints, referencePoints, results);
		if (n <= std::min(1 << 18, m << 10))
			return v7::cudaCallback(k, m, n, searchPoints, referencePoints, results);
#pragma omp parallel num_threads(num_gpus)
		{
			int thread_num = omp_get_thread_num(),
				thread_n = divup(n, num_gpus);
			float *thread_referencePoints = referencePoints + thread_num * thread_n * k;
			if (thread_num == num_gpus - 1)
			{
				thread_n = n - thread_num * thread_n;
				if (thread_n == 0)
					thread_n = 1, thread_referencePoints -= k;
			}
			CHECK(hipSetDevice(thread_num));
			thrust::device_vector<float>
				s_d(searchPoints, searchPoints + k * m),
				r_d(k * thread_n);
			{
				thrust::device_vector<float>
					rr_d(thread_referencePoints,
						 thread_referencePoints + k * thread_n);
				const int BLOCK_DIM_X = 32, BLOCK_DIM_Y = 32;
				//WuKTimer t1;
				mat_inv_kernel<<<
					dim3(divup(thread_n, BLOCK_DIM_X), divup(k, BLOCK_DIM_Y)),
					dim3(BLOCK_DIM_X, BLOCK_DIM_Y)>>>(
					k,
					thread_n,
					thrust::raw_pointer_cast(rr_d.data()),
					thrust::raw_pointer_cast(r_d.data()));
			}
			const int BLOCK_DIM_X = 1024;
			int numBlocks;
			CHECK(hipOccupancyMaxActiveBlocksPerMultiprocessor(
				&numBlocks,
				cudaCallbackKernel<BLOCK_DIM_X>,
				BLOCK_DIM_X,
				0));
			thrust::device_vector<int> results_d(m * divup(numBlocks, m));
			{
				//WuKTimer t2;
				cudaCallbackKernel<
					BLOCK_DIM_X><<<
					dim3(results_d.size() / m, m),
					BLOCK_DIM_X>>>(
					k,
					m,
					thread_n,
					results_d.size(),
					thrust::raw_pointer_cast(s_d.data()),
					thrust::raw_pointer_cast(r_d.data()),
					thrust::raw_pointer_cast(results_d.data()));
			}
			int my_beg, my_end;
#pragma omp critical
			{
				my_beg = results_tmp.size();
				results_tmp.insert(results_tmp.end(), results_d.begin(), results_d.end());
				my_end = results_tmp.size();
			}
#pragma omp barrier
			for (int offset = (thread_referencePoints - referencePoints) / k; my_beg < my_end; ++my_beg)
				results_tmp[my_beg] += offset;
		}
		*results = (int *)malloc(sizeof(int) * m);
#pragma omp parallel for
		for (int mInd = 0; mInd < m; ++mInd)
		{
			float minSquareSum = INFINITY;
			int minIndex = 0;
			// Iterate over all reference points
			for (int i = 0; i < results_tmp.size(); i += m)
			{
				const int nInd = results_tmp[i];
				float squareSum = 0;
				for (int kInd = 0; kInd < k; ++kInd)
				{
					const float diff = searchPoints[k * mInd + kInd] - referencePoints[k * nInd + kInd];
					squareSum += diff * diff;
				}
				if (minSquareSum > squareSum)
				{
					minSquareSum = squareSum;
					minIndex = nInd;
				}
			}
			(*results)[mInd] = minIndex;
		}
	}
}; // namespace v8
struct WarmUP
{
	WarmUP(int k, int m, int n)
	{
		void (*cudaCallback[])(int, int, int, float *, float *, int **) = {
			v0::cudaCallback,
			v1::cudaCallback,
			v2::cudaCallback,
			v3::cudaCallback,
			v4::cudaCallback,
			v5::cudaCallback,
			v6::cudaCallback,
			v7::cudaCallback,
			v8::cudaCallback}; //由于多卡版本是调用单卡版本实现的，因此无需热身
		float *searchPoints = (float *)malloc(sizeof(float) * k * m);
		float *referencePoints = (float *)malloc(sizeof(float) * k * n);

#pragma omp parallel
		{
			unsigned seed = omp_get_thread_num(); //每个线程使用不同的随机数种子
#pragma omp for
			for (int i = 0; i < k * m; ++i)
				searchPoints[i] = rand_r(&seed) / double(RAND_MAX); //使用线程安全的随机数函数
#pragma omp for
			for (int i = 0; i < k * n; ++i)
				referencePoints[i] = rand_r(&seed) / double(RAND_MAX);
		}

		for (int i = 0; i < sizeof(cudaCallback) / sizeof(cudaCallback[0]); ++i)
		{
			int *result;
			cudaCallback[i](k, m, n, searchPoints, referencePoints, &result);
			free(result);
		}
		free(searchPoints);
		free(referencePoints);
	}
};
struct BenchMark
{
	BenchMark(int k, int m, int n)
	{
		void (*cudaCallback[])(int, int, int, float *, float *, int **) = {
			v0::cudaCallback,
			v1::cudaCallback,
			v2::cudaCallback,
			v3::cudaCallback,
			v4::cudaCallback,
			v5::cudaCallback,
			v6::cudaCallback,
			v7::cudaCallback,
			v8::cudaCallback}; //由于多卡版本是调用单卡版本实现的，因此无需热身
		float *searchPoints = (float *)malloc(sizeof(float) * k * m);
		float *referencePoints = (float *)malloc(sizeof(float) * k * n);

#pragma omp parallel
		{
			unsigned seed = omp_get_thread_num(); //每个线程使用不同的随机数种子
#pragma omp for
			for (int i = 0; i < k * m; ++i)
				searchPoints[i] = rand_r(&seed) / double(RAND_MAX); //使用线程安全的随机数函数
#pragma omp for
			for (int i = 0; i < k * n; ++i)
				referencePoints[i] = rand_r(&seed) / double(RAND_MAX);
		}
		printf("\n\nStart benchmark with (k, m, n) = (%d, %d, %d):\n\n", k, m, n); //开始benchnmark
		for (int i = 0; i < sizeof(cudaCallback) / sizeof(cudaCallback[0]); ++i)
		{
			int *result;
			printf("Version %d: ", i);
			{
				WuKTimer t1;
				cudaCallback[i](k, m, n, searchPoints, referencePoints, &result);
			}
			free(result);
			fflush(stdout);
		}
		printf("\nFinish benchmark with (k, m, n) = (%d, %d, %d).\n\n", k, m, n);
		free(searchPoints);
		free(referencePoints);
	}
};
static WarmUP warm_up(1, 1, 1048576);
static BenchMark
	benchmark1(16384, 1, 65536),
	benchmark1024(16, 1024, 1048576);
void cudaCallback(
	int k,
	int m,
	int n,
	float *searchPoints,
	float *referencePoints,
	int **results)
{
	v8::cudaCallback(
		k,
		m,
		n,
		searchPoints,
		referencePoints,
		results);
}